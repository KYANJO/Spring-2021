#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#include "kernel.cu"

__global__ void test_init(float *u, float *u_new, int N)
{
  unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int j = tjreadIdx.y + blockIdx.y * blockDIm.y;

  unsigned int idx = j * N + i;

  if(i<N && j<N) {
    u[idx] = 0.0;
    u_new[idx] = 1.0;
  }
  
}

int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    float error

    // set up problem size
    int N = 6
    int nxy = N * N;
    int nBytes = nxy * sizeof(float);
    printf("Problem size: nx %d ny %d\n", N, N);
    
    // malloc device global memory
    float *d_u, *d_u_new, *d_error;
    CHECK(hipMalloc((void **)&d_u     , nBytes));
    CHECK(hipMalloc((void **)&d_u_new , nBytes));
    CHECK(hipMalloc((void **)&d_error , sizeof(float)));
    
    //set-up blocks and threads
    int dimx = 3;
    int dimy = 3;
    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    //initialize u
    
    test_init<<<grid,block>>>(d_u, d_u_new, N)
    CHECK(hipDeviceSynchronize());
    
    // compute error
    computeError<<<grid, block>>>(d_error, d_u, d_u_new, N);
    CHECK(hipDeviceSynchronize());
    
    CHECK(hipMemcpy(error, d_error, sizeof(float), hipMemcpyDeviceToHost));
    printf("Error after initialization: %e, expected value: 1.0\n",error);
    
    // update

    updateSolution<<<grid,block>>>(d_u, d_u_new, N);
    CHECK(hipDeviceSynchronize());
    
    //compute error again
    computeError<<<grid, block>>>(d_error, d_u, d_u_new, N);
    CHECK(hipDeviceSynchronize());
    
    CHECK(hipMemcpy(error, d_error, sizeof(float), hipMemcpyDeviceToHost));
    printf("Error after update: %e, expected value: 0.0\n",error);
    
    // check kernel error
    CHECK(hipGetLastError());

    // free device global memory
    CHECK(hipFree(d_u));
    CHECK(hipFree(d_u_new));
    CHECK(hipFree(d_error));

    // reset device
    CHECK(hipDeviceReset());

    return (0);
}
