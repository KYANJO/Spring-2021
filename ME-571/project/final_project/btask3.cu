#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "common.h"
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

#define THREADS_PER_BLOCK 256

__global__ void reductionOnGPU(float *f, const int N, const int nb, hiprandState *states)
{   
	__shared__ double u[THREADS_PER_BLOCK];
	//extern __shared__ float u[];
	int i_glb = blockIdx.x * blockDim.x + threadIdx.x; 
	unsigned int i_loc = threadIdx.x;
	int ib = blockDim.x; 
	//unsigned int i_locb = blockIdx.x;
	unsigned int i;

	int n = N/nb;
	//double cc = 0;
	//initialse hiprand
	hiprand_init(1234, i_glb, 0, &states[i_glb]);
	//hiprandState state = states[ix];

	if (i_loc<n)
	{
		double xran = hiprand_uniform_double (&states[i_glb]);
		u[i_loc] = cos(-log(xran));
		__syncthreads();
		//u[i_loc] = cc;
	}
	//u[i_loc] = cc;
	//__syncthreads();
	
	for (i = 1; i<ib; i *=2)	
	{
		__syncthreads();
		if(i_loc % (2*i) == 0)
		{
			u[i_loc] += u[i + i_loc];
		}
	}
	__syncthreads();
	//Compute a global sum
	if(i_loc==0)
	{
		//f[blockIdx.x] = u[0];
		atomicAdd(f,u[0]); 
		//printf("%f\n", u[0]);
	}
	__syncthreads();
	//printf("%f\n", u[0]);
					
}

__global__ void integralOnGPU(float *f,double *Int ,const int N)
{
	*Int = abs(*f/N);
}


int main(int argc, char **argv)
{	
	// problem size
	int N = atoi(argv[1]);
	int T = THREADS_PER_BLOCK;

	//random number generator
	hiprandState *States;
	
	// malloc host memory
	double gpuRef;
	
	//start timing
	double iStart = seconds();

	// malloc device global memory
    float *d_f;
    double *d_Int;
    CHECK(hipMalloc((void **)&d_f, sizeof(double)));
    CHECK(hipMalloc((void **)&d_Int, sizeof(double)));
    
	//invoke the kernel
	//int B = ((N + T -1)/T);
	//if(B > 65535) B = 65535;
	int B = 64;
	int nb = N/(B*T);
	
	//states allocate memory
	CHECK(hipMalloc( (void **)&States, (B*T)*sizeof(hiprandState)));

	reductionOnGPU<<<B,T>>>(d_f, N, nb,States);
	CHECK(hipDeviceSynchronize());

	integralOnGPU<<<1,1>>>(d_f,d_Int ,N);
	CHECK(hipDeviceSynchronize());

	 // check kernel error
	 CHECK(hipGetLastError());

	 double iElaps_g = seconds() - iStart;
	
	// copy kernel result back to host side
    CHECK(hipMemcpy(&gpuRef, d_Int, sizeof(double), hipMemcpyDeviceToHost));

	//error achived
	double error = 0.5 - gpuRef;
	printf("%d,%f,%e,%f\n",N,gpuRef,error,iElaps_g);
	
	//free device memory
	CHECK(hipFree(States));
	CHECK(hipFree(d_f));
    CHECK(hipFree(d_Int));

    // reset device
    CHECK(hipDeviceReset());

    return (0);
}
