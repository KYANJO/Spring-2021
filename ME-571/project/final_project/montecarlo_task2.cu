#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "common.h"
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

#define THREADS_PER_BLOCK 256

__global__ void mcOnGPU(float *f, const int N, const int nb, hiprandState *states)
{   
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
	int n = N/nb;
	float cc = 0.0f;
	//initialse hiprand
	hiprand_init(1234, ix, 0, &states[ix]);
	//hiprandState state = states[ix];
	if (ix<n)
	{
		float xran = hiprand_uniform (&states[ix]);
		cc += cos(-log(xran));
		
		//__syncthreads();

		atomicAdd(f,cc); 
		__syncthreads();
	}
					
}

__global__ void integralOnGPU(float *f,float *Int ,const int N)
{
	Int[0] = abs(f[0]/N);
}


int main(int argc, char **argv)
{	
	// problem size
	int N = atoi(argv[1]);
	int T = THREADS_PER_BLOCK;
	
	//random number generator
	hiprandState *States;
	
	// malloc host memory
    float gpuRef;
	
	// malloc device global memory
    float *d_f;
    float *d_Int;
    CHECK(hipMalloc((void **)&d_f, sizeof(float)));
    CHECK(hipMalloc((void **)&d_Int, sizeof(float)));
    
	//invoke the kernel
	int B = ((N + T -1)/T);
	if(B > 65535) B = 65535;
	int nb = N/(B*T);
	
	//states allocate memory
	CHECK(hipMalloc( (void **)&States, (B*T)*sizeof(hiprandState)));

	mcOnGPU<<<B,T>>>(d_f, N, nb, States);
	CHECK(hipDeviceSynchronize());

	integralOnGPU<<<1,1>>>(d_f,d_Int ,N);
	CHECK(hipDeviceSynchronize());

	// copy kernel result back to host side
    CHECK(hipMemcpy(&gpuRef, d_Int, sizeof(float), hipMemcpyDeviceToHost));

	//error achived
	float error = 0.5 - gpuRef;
	printf("%d,%f,%e\n",N,gpuRef,error);
	
	//free device memory
	CHECK(hipFree(States));
	CHECK(hipFree(d_f));
    CHECK(hipFree(d_Int));

    // reset device
    CHECK(hipDeviceReset());

    return (0);
}
