#include "hip/hip_runtime.h"
/******************************************************************************
 * Implements the Monte-Carlo method of performing numerical integral for complex
 * integrals on the GPU
 * 
 * Author: Yao GAhounzo
 * Date : 04/21/2021
 */


 #include <hiprand.h>
 #include <hiprand/hiprand_kernel.h>
 #include "common.h"
 #include <hip/hip_runtime.h>
 #include <stdio.h>
 #include <math.h>
 
 #define THREADS_PER_BLOCK 64

 // performs the montecarlo integration on the GPU
 
 __global__ void monteCarloOnGPU(hiprandState *state, float *fx, float *Integ, const int N, const int BT){	
     
    __shared__ float partialSum[THREADS_PER_BLOCK];
    //__shared__ float sumThread;
    float sumThread = 0;
    double x;
    
    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int id_loc = threadIdx.x;

    int Nx = N/BT;
     
    if(id < Nx){	

		// Initialize CURAND
		hiprand_init(id, id, 0, &state[id]);

		for(int i = 0; i < BT; i++){

			x  = hiprand_uniform(&state[id]);
			sumThread += cos(-log(x));
			 
		}

    }      
    
    //__syncthreads();
    partialSum[id_loc] = sumThread;
    __syncthreads();
    
    // Reduction in shared memory
    for (unsigned int i = 1; i < blockDim.x; i *= 2){
        if (id_loc % (2*i) == 0) {
            partialSum[id_loc] += partialSum[id_loc + i];
        }
        __syncthreads();
    }
    
    //__syncthreads();
    if (id_loc == 0) fx[blockIdx.x] = partialSum[0];


 }
 
 
 __global__ void Integral(float *fx, float *Integ, const int N){

     int id = blockIdx.x;
    __syncthreads();
 
    atomicAdd(Integ, fx[id]);
     
   // __syncthreads();
 
    *Integ = *Integ / N; 

 }
 
 int main(int argc, char **argv){
     
     printf("\n");
     printf("%s Starting...\n", argv[0]);
 
     // set up device
     int dev = 0;
     hipDeviceProp_t deviceProp;
     CHECK(hipGetDeviceProperties(&deviceProp, dev));
     printf("Using Device %d: %s\n\n", dev, deviceProp.name);
     CHECK(hipSetDevice(dev));
 
 
     int N = atoi(argv[1]);
     
     double iStart, iElaps, iStart_comp, iElaps_comp;
     iStart = seconds();
     // host memory
     //int size = sizeof(float);
     float gpuRef,  error;
     float exact = 0.5;
 
     // malloc device memory
     float *Integ_d, *fx_d;
     CHECK(hipMalloc((float **)&Integ_d, sizeof(float)));
     //CHECK(hipMalloc((float **)&fx_d, sizeof(float)));
 
     // invoke kernel
     //int nThreads = 64;
     int nThreads = THREADS_PER_BLOCK;
     int nBlocks = ((N + nThreads - 1)/nThreads);
     //int nBlocks = N/nThreads;
 
     if(nBlocks > 65535){
         nBlocks = 65535;
     }
     
     int BT = N/(nThreads*nBlocks);
     if(BT == 0) BT = 1;
     int Nx = N/BT;
 
     hiprandState *devStates;
     //float *random_d;
     CHECK(hipMalloc((void **)&devStates, Nx*sizeof(hiprandState)));
     //CHECK(hipMalloc((void **)&random_d, Nx*sizeof(float)));
     CHECK(hipMalloc((float **)&fx_d, N*sizeof(float)));
 
     // On GPU
     iStart_comp = seconds();
     monteCarloOnGPU<<<nBlocks, nThreads>>>(devStates, fx_d, Integ_d, N, BT);
     CHECK(hipDeviceSynchronize());
     iElaps_comp = seconds() - iStart_comp;
     
     iStart_comp = seconds();
     Integral<<<2, 1024>>>(fx_d, Integ_d, N);
     CHECK(hipDeviceSynchronize());
     iElaps_comp = seconds() - iStart_comp;
     
     // check kernel error
     CHECK(hipGetLastError());
 
     // copy kernel result back to host side
     CHECK(hipMemcpy(&gpuRef, Integ_d, sizeof(float), hipMemcpyDeviceToHost));
 
     // compute error
     error = abs(exact - gpuRef);
 
     iElaps = seconds() - iStart;
 
     printf("N = %ld, Integral = %f, error = %e, elapsed_time = %f s, time_comp = %f s\n\n",N,gpuRef,error,iElaps,iElaps_comp);
 
     // free device global memory
     CHECK(hipFree(Integ_d));
     CHECK(hipFree(fx_d));
 
     // reset device
     CHECK(hipDeviceReset());
     
     return 0;
 }
 