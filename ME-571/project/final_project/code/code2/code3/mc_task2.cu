#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "common.h"
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

#define THREADS_PER_BLOCK 64

__global__ void mcOnGPU(float *f, const int N, const int nb, hiprandState *states)
{   
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
	int n = N/nb;
	
	double cc = 0;
	//initialse hiprand
	hiprand_init(1234, ix, 0, &states[ix]);
	//hiprandState state = states[ix];
	if (ix<n)
	{	
		for(int i=0; i<nb; i++)
		{
			double xran = hiprand_uniform_double (&states[ix]);
			cc += cos(-log(xran));
		}
	}
		atomicAdd(f,cc); 
		__syncthreads();				
}

__global__ void integralOnGPU(float *f,double *Int ,const int N)
{
	Int[0] = abs(f[0]/N);
}


int main(int argc, char **argv)
{	
	// problem size
	long long int N = atol(argv[1]);
	int T = THREADS_PER_BLOCK;
	
	//random number generator
	hiprandState *States;
	
	// malloc host memory
	double gpuRef;
	
	//start timing
	double iStart = seconds();

	// malloc device global memory
    float *d_f;
    double *d_Int;
    CHECK(hipMalloc((void **)&d_f, sizeof(double)));
    CHECK(hipMalloc((void **)&d_Int, sizeof(double)));
    
	//invoke the kernel
	int B = ((N + T -1)/T);
	if(B > 65535) B = 65535;
	int nb = ceil((N*1.0)/(B*T));
	
	//states allocate memory
	CHECK(hipMalloc( (void **)&States, (B*T)*sizeof(hiprandState)));

	mcOnGPU<<<B,T>>>(d_f, N, nb, States);
	CHECK(hipDeviceSynchronize());

	integralOnGPU<<<1,1>>>(d_f,d_Int ,N);
	CHECK(hipDeviceSynchronize());

	 // check kernel error
	CHECK(hipGetLastError());

	double iElaps_g = seconds() - iStart;
	
	// copy kernel result back to host side
    CHECK(hipMemcpy(&gpuRef, d_Int, sizeof(double), hipMemcpyDeviceToHost));

	//error achived
	double error = 0.5 - gpuRef;
	printf("%lld,%f,%e,%f\n",N,gpuRef,error,iElaps_g);
	
	//free device memory
	CHECK(hipFree(States));
	CHECK(hipFree(d_f));
    CHECK(hipFree(d_Int));

    // reset device
    CHECK(hipDeviceReset());

    return (0);
}
