#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "common.h"
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

#define THREADS_PER_BLOCK 256

__global__ void mcrandom(double *udata, const int N, const int nb, hiprandState *states)
{
	unsigned int i_glb = blockIdx.x * blockDim.x + threadIdx.x; 
	int n = N/nb;
	//initialse hiprand
	double cc = 0;
	hiprand_init(1234, i_glb, 0, &states[i_glb]);
	if (i_glb<n)
	{
		double xran = hiprand_uniform_double (&states[i_glb]);
		cc += cos(-log(xran));
	}
	udata[i_glb] = cc;
}

__global__ void reductionOnGPU(double *udata, float *f, float *c)
{   
	__shared__ double u[THREADS_PER_BLOCK];
	//extern __shared__ double u[];
	unsigned int i_glb = blockIdx.x * blockDim.x + threadIdx.x; 
	unsigned int i_loc = threadIdx.x;
	int ib = blockDim.x; 
	unsigned int i;

	//load memory
	u[i_loc] = udata[i_glb];
	__syncthreads();

	/*
	//reduction in shared memory
	for (i = 1; i<ib; i *=2)	
	{
		if(i_loc % (2*i) == 0)
		{
			//__syncthreads();
			u[i_loc] += u[i + i_loc];
		}
		__syncthreads();
	}
*/

	for (i = 1; i<ib; i *=2)	
	{
		int index = 2*i*i_loc;
		if (index < blockDim.x) 
		{
			u[index] += u[i + index];
		}
		__syncthreads();
	}
	
	//append top the a global memory
	if(i_loc==0)
	{
		c[blockIdx.x] = u[0];
		//printf("%f\n",*f);
	}
	//printf("%f\n",*c);
	//atomicAdd(f,c);

	
	/*
	for (i = 1; i<ib; i *=2)	
	{
		int index = 2*i*i_loc;
		__syncthreads();
		if (index < blockDim.x) 
		{
			c[index] += c[i + index];
		}
		__syncthreads();
	}
		
	//if(i_loc==0)
	//{
		f[blockIdx.x] = c[0];
		//printf("%f\n",*f);
	//}
    */
}

__global__ void integralOnGPU(float *f,double *Int ,const int N)
{
	*Int = abs(*f/N);
}


int main(int argc, char **argv)
{	
	// problem size
	int N = atoi(argv[1]);
	int T = THREADS_PER_BLOCK;

	//random number generator
	hiprandState *States;
	
	// malloc host memory
	double gpuRef;
	
	//start timing
	double iStart = seconds();

	// malloc device global memory
	float *d_f;
	double *d_Int;
	double *d_udata;
	float *d_c;
    CHECK(hipMalloc((void **)&d_f, sizeof(double)));
	CHECK(hipMalloc((void **)&d_Int, sizeof(double)));
	CHECK(hipMalloc((void **)&d_udata, T));
	CHECK(hipMalloc((void **)&d_c, T));
	
	//invoke the kernel
	//int B = ((N + T -1)/T);
	//if(B > 65535) B = 65535;
	int B = 64;
	int nb = N/(B*T);
	
	//states allocate memory
	CHECK(hipMalloc( (void **)&States, (B*T)*sizeof(hiprandState)));
    
	mcrandom<<<B,T>>>(d_udata, N, nb, States);
	CHECK(hipDeviceSynchronize());
	reductionOnGPU<<<B,T>>>(d_udata,d_f,d_c);
	CHECK(hipDeviceSynchronize());

	integralOnGPU<<<1,1>>>(d_f,d_Int ,N);
	CHECK(hipDeviceSynchronize());

	 // check kernel error
	CHECK(hipGetLastError());

	 double iElaps_g = seconds() - iStart;
	
	// copy kernel result back to host side
    CHECK(hipMemcpy(&gpuRef, d_Int, sizeof(double), hipMemcpyDeviceToHost));

	//error achived
	double error = 0.5 - gpuRef;
	printf("%d,%f,%e,%f\n",N,gpuRef,error,iElaps_g);
	
	//free device memory
	CHECK(hipFree(States));
	CHECK(hipFree(d_f));
    CHECK(hipFree(d_Int));
	CHECK(hipFree(d_udata));
	CHECK(hipFree(d_c));

    // reset device
    CHECK(hipDeviceReset());

    return (0);
}
