#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * This example demonstrates a matrix-matrix multiplication on the CPU.
 */

 void initialData(float *ip, const float ival, int size)
 {
     for (int i = 0; i < size; i++)
     {
         ip[i] = (float)(rand() & 0xFF) / 100.0f;
     }
 
     return;
 }

void matmulOnHost(float *A, float *B, float *C, const int N)
{
  int   id, ida, idb;
  float cc;
  
    for (int iy = 0; iy < N; iy++)
    {
        for (int ix = 0; ix < N; ix++)
        {
            
            cc = 0;
            for (int k = 0; k < N; k++){
                ida = iy*N + k;
                idb = k *N + ix;
                    cc += A[ida]*B[idb];
	        }
	  id = iy*N+ix;
	  C[id] = cc;
        }
    }

    return;
}
// grid 2D block 2D
__global__ void matmulOnGPU(float *A, float *B, float *C, const int N)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx;
    unsigned int ida, idb;

    float cc = 0.0f;

    if (ix < N && iy < N)
    {
        for (int k = 0; k < N; k++)
        {
            ida = iy*N + k;
            idb = k *N + ix;
            cc += A[ida]*B[idb];
        }
    }
    idx = iy * N + ix;
    C[idx] = cc;
}
  
void printMatrix(float *C, const int nx, const int ny)
{
    float *ic = C;

    for (int iy = 0; iy < ny; iy++)
    {
        for (int ix = 0; ix < nx; ix++)
        {
            printf("%f ", ic[ix]);

        }

        ic += nx;
        printf("\n");
    }

    return;
}
void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-6;
    bool match = 1;
    for (int i = 0; i < N; i++)
    {
      if (abs(hostRef[i] - gpuRef[i])/abs(hostRef[i]) > epsilon)
        {
            match = 0;
            printf("host %f gpu %f, err = %e\n", hostRef[i], gpuRef[i], abs(hostRef[i]-gpuRef[i])/abs(hostRef[i]));
            break;
        }
    }
    if (match)
        printf("Arrays match.\n\n");
    else
        printf("Arrays do not match.\n\n");
}

int main(int argc, char* argv[])
{

    // set up data size of matrix
    int N = 1 << 11;
   
    int nxy = N * N;
    int nBytes = nxy * sizeof(float);
    //printf("Matrix size: nx %d ny %d\n", N, N);

    // malloc host memory
    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);


    // initialize data at host side
    initialData(h_A,2.0f,nxy);
    initialData(h_B,0.5f, nxy);

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    double iStart = seconds();
    matmulOnHost(h_A, h_B, hostRef, N);
    double iElaps_h = seconds() - iStart;
    //printf("matmul elapsed %f sec\n", iElaps);

    // malloc device global memory
    float *d_A, *d_B, *d_C;
    CHECK(hipMalloc((void **)&d_A, nBytes));
    CHECK(hipMalloc((void **)&d_B, nBytes));
    CHECK(hipMalloc((void **)&d_C, nBytes));

    // transfer data from host to device
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
    //CHECK(hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice));

    // invoke kernel at host side
    int nb = atol(argv[1]); //block size

    int dimx = nb;
    int dimy = nb;
    dim3 block(dimx, dimy);
    dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);

    iStart = seconds();
    matmulOnGPU<<<grid, block>>>(d_A, d_B, d_C, N);
    CHECK(hipDeviceSynchronize());
    double iElaps_g = seconds() - iStart;
    //printf("matmulOnGPU <<<(%d,%d), (%d,%d)>>> elapsed %f sec\n", grid.x,
          // grid.y,
           //block.x, block.y, iElaps);
    printf("%d,%d,%f,%f\n",N,nb, iElaps_h,iElaps_g);
    // check kernel error
    CHECK(hipGetLastError());

    // copy kernel result back to host side
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    // check device results
    //checkResult(hostRef, gpuRef, nxy);

    // free device global memory
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));

    // free host memory
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    // reset device
    CHECK(hipDeviceReset());

    return (0);
}
