#include "hip/hip_runtime.h"
/***************************
finite_difference computes an approximation to derivative of a function
using 2nd order finite difference method

dudx = (u(x+dx) - u(x-dx))/(2*dx)

Written by: Michal A. Kopera
            Department of Mathematics
            Boise State University
            1/12/2021

Based on Example 6.6 in Chopp, D.L. 'Introduction to High Performance Scientific Computing"
 **************************/
#include <stdio.h>
#include "common.h"
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 1024

/*
diff comoutes the approximation to the derivative dudx to function u given by a set of discrete points

Inputs:
u  - an array of values of function u at discrete points
N  - number of points in the u array
dx - distance between points (assunes equidistant distribution)

Outputs:
dudx - contains the finite difference approximation 
*/

void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i],
                   gpuRef[i], i);
	    // break;
        }
    }

if (match) printf("Arrays match.\n\n");

    return;
}



void diffOnHost(float *u,const int N, float dx,float *dudx )
{
 int i; // local index for traversing arrays

  //at the left end-point, use one-sided difference
  dudx[0] = (u[1] - u[0])/dx; 

  // go over interior points and compute second-order finite difference
  for (i=1; i<N-1; ++i)
    {
      dudx[i] = (u[i+1] - u[i-1])/dx/2.0;
    }

  //at the right end-point compute one-sided difference
  dudx[N-1] = (u[N-1] - u[N-2])/dx; 
}


__global__ void diffOnGPU(float *u, const int N, float dx, float *dudx )
{
 int i = blockIdx.x * blockDim.x + threadIdx.x; 


  if(i==0)   //at the left end-point, use one-sided difference
  {
    dudx[i] = (u[i+1] - u[i])/dx;
    }
  else if(i==N-1)   //at the right end-point compute one-sided difference
  {
    dudx[i] = (u[i] - u[i-1])/dx;
  }
    else   // go over interior points and compute second-order finite difference
    {
      dudx[i] = (u[i+1] - u[i-1])/dx/2.0;
    }

}

__global__ void diffOnGPU_shared(float *u, const int N, float dx, float *dudx )
{

  float u_shared[THREADS_PER_BLOCK]; //declare shared array
 int i_glb = blockIdx.x * blockDim.x + threadIdx.x; 
 int i_loc = threadIdx.x;
 
 //load data into shared array
 if(i_glb<N){
   u_shared[i_loc] = u[i_glb];
 }
 
 __syncthreads(); //make sure copying to shared array is done
 
 //operate on shared array instead
  if(i_glb==0)   //at the global left end-point, use one-sided difference
  {
    dudx[i_glb] = (u[i_glb+1] - u[i_glb])/dx;
    }
  else if(i_glb==N-1)   //at the global right end-point compute one-sided difference
  {
    dudx[i_glb] = (u[i_glb] - u[i_glb-1])/dx;
  }
    else   // go over interior points and compute second-order finite difference
    {
      if(i_loc>0 && i_loc < THREADS_PER_BLOCK-1) { //if not the first and last thread in a block
	dudx[i_glb] = (u_shared[i_loc+1] - u_shared[i_loc-1])/dx/2.0;
      }else{
	dudx[i_glb] = (u[i_glb+1] - u[i_glb-1])/dx/2.0;
      }
    }

}

__global__ void diffOnGPU_shared1(float *u, const int N, float dx, float *dudx )
{

  float u_shared[THREADS_PER_BLOCK+2]; //declare shared array including ghost points
 int i_glb = blockIdx.x * blockDim.x + threadIdx.x; 
 int i_loc = threadIdx.x+1; //shift the local index to account for ghost points
 
 //load data into shared array
 if(i_glb<N){
   u_shared[i_loc] = u[i_glb];
 }

 if(threadIdx.x==0){
   u_shared[0] = u[i_glb-1]; //first thread loads the ghost from neighboring left block into ghost
 }

 if(threadIdx.x==THREADS_PER_BLOCK-1 && i_glb<N){
   u_shared[THREADS_PER_BLOCK+1] = u[i_glb+1]; //last thread loads the ghost from neighboring right block into ghost
 }
 
 __syncthreads(); //make sure copying to shared array is done
 
 //operate on shared array
 if(i_glb==0)   //at the global left end-point, use one-sided difference
   {
     dudx[i_glb] = (u[i_glb+1] - u[i_glb])/dx;
   }
 else if(i_glb==N-1)   //at the global right end-point compute one-sided difference
   {
     dudx[i_glb] = (u[i_glb] - u[i_glb-1])/dx;
   }
 else   // go over interior points and compute second-order finite difference
   {
     dudx[i_glb] = (u_shared[i_loc+1] - u_shared[i_loc-1])/dx/2.0;
   }
 
}


/*
init_u initializes array u with some data
Here we use a sin function

Inputs:
N  - the number of points
dx - the distance between points
 
Outputs:
u  - the values of u at specified points
*/

void init_u(float *u, double dx, const int N)
{
 int i;

  //for each point compute the value of our function at i*dx
  for (i=0; i<N; ++i)
    {
      u[i] = sin(i*dx);
    }
}

__global__ void initOnGPU(float *u, double dx, const int N)
{

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i<N)
    {
        u[i] = sin(i*dx);
    }
}

/* main function 
Creates the function which derivative is computed and measures the execution time of the finite difference computation. 

Assumes that it receives N as the first (after the executable name) command line argument
Inputs:
N - number og points used to approximate function u

*/

int main(int argc, char* argv[])
{

// set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));


  // get the number of points from input parameters
  int N = atoi(argv[1]);
  
  double istart, iElaps, istarth, iElapsh, iElap_copy, iElaps_shared; 
  
  istart = seconds();
  
  //allocate host memory
  size_t nBytes = N * sizeof(float);
  
  float *h_u,  *hostRef, *gpuRef;

  h_u     = (float *)malloc(nBytes);
  hostRef = (float *)malloc(nBytes);
  gpuRef  = (float *)malloc(nBytes);
  
  memset(hostRef, 0, nBytes);
  memset(gpuRef,  0, nBytes);
  
  
    // malloc device global memory
   float *d_u,*d_dudx;
   CHECK( hipMalloc((float**)&d_u,nBytes));
   CHECK( hipMalloc((float**)&d_dudx,nBytes));
  
  
  
  // compute the interval size dx; M_PI holds value of pi
  double dx = 2.0*M_PI/(N-1);



  //create initial condition on host
  init_u(h_u,dx,N);

  istarth = seconds();
  
    //call finite difference function on host
  diffOnHost(h_u,N,dx,hostRef);
 
 iElapsh = seconds() - istarth;
 
 
 // invoke kernel at host side
int iLen = THREADS_PER_BLOCK;
 dim3 block (iLen); //how many threads in a block
 dim3 grid  ((N + block.x -1) / block.x);


 
initOnGPU<<<grid,block>>>(d_u,dx,N);
CHECK(hipDeviceSynchronize());
  /*
int nBlocks = 1;
 int nThreads = 1024; */


//time the global memory kernel
  istart = seconds();
  diffOnGPU<<<grid, block>>>(d_u,N,dx,d_dudx);
 CHECK(hipDeviceSynchronize());
//diffOnGPU<<<nBlocks, nThreads>>>(d_u,N,dx,d_dudx);
 // copy kernel result back to host side

iElaps = seconds() - istart;

CHECK(hipMemcpy(gpuRef, d_dudx, nBytes, hipMemcpyDeviceToHost));
// check device results
 checkResult(hostRef, gpuRef, N);


 //time the shared memory kernel
 istart = seconds();
 //diffOnGPU_shared<<<grid, block>>>(d_u,N,dx,d_dudx);
 CHECK(hipDeviceSynchronize());
 iElaps_shared = seconds() - istart;


 
 istart = seconds();
 
CHECK(hipMemcpy(gpuRef, d_dudx, nBytes, hipMemcpyDeviceToHost));
 
iElap_copy = seconds() - istart;
 
// check kernel error
    CHECK(hipGetLastError()) ;
 
  // check device results
    checkResult(hostRef, gpuRef, N);


    printf("N = %d, threads = %d, blocks = %d,  timelapsed host = %f device_global = %f device_local = %f copy = %f\n ",N,block.x,grid.x,iElapsh,iElaps,iElaps_shared,iElap_copy);
  
  
  //free device global memory
 CHECK( hipFree(d_u));
 CHECK( hipFree(d_dudx));
  
  //free host memory
  free(h_u);
  free(hostRef);
  free(gpuRef);
  
  hipDeviceReset();
 
 return(0);
}
