//Just your regular Hello World file
// to be compiled with nvcc rather than gcc


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void) {
  printf("Hello World from GPU, block %d, thread %d\n",blockIdx.x,threadIdx.x);
}


int main(void) {
  printf("Hello World from CPU!\n");

  helloFromGPU<<<1, 10>>>();
  hipDeviceReset();

  return 0;
}
